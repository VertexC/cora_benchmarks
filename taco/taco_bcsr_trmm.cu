#include "hip/hip_runtime.h"
#include <iostream>
#include "taco.h"
#include "utils.cuh"

using namespace taco;
using namespace std::chrono;

const IndexVar io("io"), jo("jo"), ko("ko"), ii("ii"), ji("ji"), ki("ki");
int WARP_SIZE = 32;

IndexStmt scheduleSpMMGPU(IndexStmt stmt, Tensor<float> A, int m, int bs, IndexExpr precomputedAExpr,
			  int NNZ_PER_WARP=1, int BLOCK_SIZE=256) {
  int NNZ_PER_TB = NNZ_PER_WARP * (BLOCK_SIZE / WARP_SIZE);
  IndexVar f("f"), fpos("fpos"), block("block"), fpos1("fpos1"), warp("warp"), nnz("nnz"), nnz_pre("nnz_pre");
  IndexVar dense_val_unbounded("dense_val_unbounded"), dense_val("dense_val"), thread("thread");
  IndexVar thread_nz("thread_nz");
  TensorVar precomputedA("precomputedA", Type(Float32, {Dimension(nnz)}), taco::dense);
  return stmt.reorder({io, jo, ko, ii, ji, ki})
  .fuse(io, jo, f)
    .pos(f, fpos, A(io, jo, ii, ji))
    .split(fpos, block, fpos1, NNZ_PER_TB)
    .split(fpos1, warp, nnz, NNZ_PER_WARP)
    .split(ko, dense_val_unbounded, thread, WARP_SIZE)
    .reorder({block, warp, thread, dense_val_unbounded, nnz})
    .bound(dense_val_unbounded, dense_val, (m / bs) / WARP_SIZE, BoundType::MaxExact)
    .unroll(dense_val, 4)
    .parallelize(block, ParallelUnit::GPUBlock, OutputRaceStrategy::IgnoreRaces)
    .parallelize(warp, ParallelUnit::GPUWarp, OutputRaceStrategy::IgnoreRaces)
    .parallelize(thread, ParallelUnit::GPUThread, OutputRaceStrategy::Atomics);
}

__global__
void computeDeviceKernel0(taco_tensor_t * __restrict__ A, taco_tensor_t * __restrict__ B, taco_tensor_t * __restrict__ C, int32_t* io_blockStarts, int mb){
  int A1_dimension = (int)(A->dimensions[0]);
  int A3_dimension = (int)(A->dimensions[2]);
  int A4_dimension = (int)(A->dimensions[3]);
  int* __restrict__ A2_pos = (int*)(A->indices[1][0]);
  int* __restrict__ A2_crd = (int*)(A->indices[1][1]);
  float* __restrict__ A_vals = (float*)(A->vals);
  int B2_dimension = (int)(B->dimensions[1]);
  int B3_dimension = (int)(B->dimensions[2]);
  int B4_dimension = (int)(B->dimensions[3]);
  float* __restrict__ B_vals = (float*)(B->vals);
  int C2_dimension = (int)(C->dimensions[1]);
  int C3_dimension = (int)(C->dimensions[2]);
  int C4_dimension = (int)(C->dimensions[3]);
  float* __restrict__ C_vals = (float*)(C->vals);

  int32_t block = blockIdx.x;
  int32_t thread = (threadIdx.x % (32));
  int32_t warp = (threadIdx.x / 32);
  if (threadIdx.x >= 256) {
    return;
  }

  float Cl[32];
  for (int i = 0; i < 32; ++i) {
    Cl[i] = 0;
  }

  #pragma unroll 4
  for (int32_t dense_val = 0; dense_val < mb/32; dense_val++) {
    int32_t ko = dense_val * 32 + thread;
    int32_t pA2_begin = io_blockStarts[block];
    int32_t pA2_end = io_blockStarts[(block + 1)];
    int32_t fposA = block * 8 + warp;
    int32_t io_pos = taco_binarySearchBefore(A2_pos, pA2_begin, pA2_end, fposA);
    int32_t io = io_pos;
    for (int32_t nnz = 0; nnz < 1; nnz++) {
      int32_t fposA = block * 8 + warp;
      if (fposA >= A2_pos[A1_dimension])
        break;

      int32_t f = A2_crd[fposA];
      int32_t koB = f * B2_dimension + ko;
      while (fposA == A2_pos[(io_pos + 1)]) {
        io_pos = io_pos + 1;
        io = io_pos;
      }
      int32_t koC = io * C2_dimension + ko;
      for (int32_t ii = 0; ii < A3_dimension; ii++) {
        int32_t iiC = koC * C3_dimension + ii;
        int32_t iiA = fposA * A3_dimension + ii;
        for (int32_t ji = 0; ji < A4_dimension; ji++) {
          int32_t jiB = koB * B3_dimension + ji;
          int32_t jiA = iiA * A4_dimension + ji;
          for (int32_t ki = 0; ki < B4_dimension; ki++) {
            int32_t kiC = iiC * C4_dimension + ki;
            int32_t kiB = jiB * B4_dimension + ki;
	    Cl[ji] += B_vals[kiB] * A_vals[jiA];
          }
        }
	for (int32_t ki = 0; ki < B4_dimension; ki++) {
	  int32_t kiC = iiC * C4_dimension + ki;
	  C_vals[kiC] = Cl[ki];
	}
      }
    }
  }

}

float compute(taco_tensor_t *C, taco_tensor_t *B, taco_tensor_t *A, int m, int bs, int iters) {
  int A1_dimension = (int)(A->dimensions[0]);
  int* __restrict__ A2_pos = (int*)(A->indices[1][0]);

  int32_t* io_blockStarts = 0;
  gpuErrchk(hipMallocManaged((void**)&io_blockStarts, sizeof(int32_t) * ((A2_pos[A1_dimension] + 7) / 8 + 1)));
  gpuErrchk(hipMallocManaged((void**)&(C->vals), sizeof(float) * m * m));
  io_blockStarts = taco_binarySearchBeforeBlockLaunch(A2_pos, io_blockStarts, (int32_t) 0, A1_dimension, (int32_t) 8, (int32_t) 256, ((A2_pos[A1_dimension] + 7) / 8));

  int num_blocks = (A2_pos[A1_dimension] + 7) / 8;
  std::cout << "NB " << num_blocks << " " << A2_pos[A1_dimension] << std::endl;
  hipEvent_t start, end;
  float elapsed;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);

  for (int i = 0; i < iters; ++i) {
    computeDeviceKernel0<<<num_blocks, (32 * 8)>>>(A, B, C, io_blockStarts, m/bs);
  }
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed, start, end);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  hipFree(io_blockStarts);
  return elapsed;
}



int main(int argc, char* argv[]) {
  int m = std::atoi(argv[1]);
  int bs = std::atoi(argv[2]);
  int mb = m/bs;
  Tensor<float> A("A", {mb, mb, bs, bs}, {Dense, Compressed, Dense, Dense});
  Tensor<float> B("B", {mb, mb, bs, bs}, {Dense, Dense, Dense, Dense});
  Tensor<float> C("C", {mb, mb, bs, bs}, {Dense, Dense, Dense, Dense});

  for (int i = 0; i < mb; ++i) {
    for (int j = 0; j < i + 1; ++j) {
      for (int ii = 0; ii < bs; ++ii) {
  	for (int ji = 0; ji < bs; ++ji) {
  	  float rand_float = (float)rand()/(float)(RAND_MAX);
  	  A.insert({i, j, ii, ji}, rand_float);
  	}
      }
    }
    for (int j = i + 1; j < mb; j++) {
      for (int ii = 0; ii < bs; ++ii) {
  	for (int ji = 0; ji < bs; ++ji) {
  	  float rand_float = (float)rand()/(float)(RAND_MAX);
  	  B.insert({i, j, ii, ji}, rand_float);
  	  C.insert({i, j, ii, ji}, rand_float);
  	}
      }
    }
  }
  A.pack();
  B.pack();

  auto At = A.getTacoTensorT();
  auto Bt = B.getTacoTensorT();
  auto Ct = C.getTacoTensorT();

  int witers = 100;
  int iters = 100;
  // Warm up
  compute(Ct, Bt, At, m, bs, witers);

  float time = compute(Ct, Bt, At, m, bs, iters);
  time /= iters;
  std::cout << "RESULTS," << time << std::endl;



  // IndexExpr precomputedA = A(io, jo, ii, ji);
  // IndexExpr precomputedB = B(jo, ko, ji, ki);
  // C(io, ko, ii, ki) += precomputedB * precomputedA;

  // IndexStmt stmt = C.getAssignment().concretize();
  // stmt = scheduleSpMMGPU(stmt, A, m, bs, precomputedA);

  // C.compile(stmt);
}

#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "utils.h"

int find_max(std::vector<int> v) {
  int max = -10000;
  for (auto e: v) {
    if (e > max) max = e;
  }
  return max;
}

float testCuBLAS(int M, int iters, int warmup) {
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);

  float* A;
  float* B;
  float* C;

  auto op_a = HIPBLAS_OP_T;

  int lda = M;
  int ldb = M;
  int ldc = M;

  CUDA_CHECK(hipMalloc((void**)&A, M * M * sizeof(float)));
  CUDA_CHECK(hipMalloc((void**)&B, M * M * sizeof(float)));
  CUDA_CHECK(hipMalloc((void**)&C, M * M * sizeof(float)));

  auto runner = [&]() {
    float time = 0;
    for (int i = 0; i < iters; ++i) {
      hipEvent_t start, end;
      float elapsed = 0;

      // Timing info
      hipEventCreate(&start);
      hipEventCreate(&end);
      hipEventRecord(start);

      const float alpha = 1.0;

      hipblasStatus_t cublas_result = hipblasStrmm(cublas_handle,
						 HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
						 op_a, HIPBLAS_DIAG_NON_UNIT,
						 M, M,
						 &alpha,
						 A, lda,
						 B, ldb,
						 C, ldc);

      hipEventRecord(end);
      hipEventSynchronize(end);
      hipEventElapsedTime(&elapsed, start, end);
      time += elapsed;
      assert(cublas_result == HIPBLAS_STATUS_SUCCESS);
    }
    return (time / iters);
  };

  if (warmup) { runner(); }
  float time = runner();

  CUDA_CHECK(hipFree((void*)A));
  CUDA_CHECK(hipFree((void*)B));
  CUDA_CHECK(hipFree((void*)C));

  return time;
}

int main(int argc, char *argv[]) {
  int M = std::stoi(argv[1]);
  int iters = std::stoi(argv[2]);
  int warmup = std::stoi(argv[3]);

  float time = testCuBLAS(M, iters, warmup);
  std::cout << "RESULTS," << time << std::endl;
}
